#include "UtilsCuda.cuh"

hipStream_t createCudaStream() {
  hipStream_t stream;
  hipStreamCreate(&stream);
  return stream;
}