#include "TimerGpu.h"

TimerGpu::TimerGpu() : started(false), stopped(false) {
  checkError(hipEventCreate(&startEvent), "creating start event");
  checkError(hipEventCreate(&stopEvent), "creating stop event");
}

TimerGpu::~TimerGpu() {
  checkError(hipEventDestroy(startEvent), "destroying start event");
  checkError(hipEventDestroy(stopEvent), "destroying stop event");
}

void TimerGpu::reset() {}
void TimerGpu::start() {
  started = true;
  stopped = false;
  checkError(hipEventRecord(startEvent, 0), "recording start event");
}

void TimerGpu::stop() {
  stopped = true;
  checkError(hipEventRecord(stopEvent, 0), "recording stop event");
  checkError(hipEventSynchronize(stopEvent), "synchronizing on stop event");
}

double TimerGpu::elapsedMilliseconds() const {
  if (!started || !stopped) {
    std::cerr << "Timer error: Start and stop must be called before getting "
                 "elapsed time."
              << std::endl;
    return 0.0f;
  }
  float milliseconds = 0.0f;
  checkError(hipEventElapsedTime(&milliseconds, startEvent, stopEvent),
             "calculating elapsed time");
  return milliseconds;
}
